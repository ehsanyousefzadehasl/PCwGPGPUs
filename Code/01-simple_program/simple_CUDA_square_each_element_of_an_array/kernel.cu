#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>

using namespace std;

__global__ void square(float* d_out, float* d_in)
{
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main()
{
    const int ARRAY_SIZE = 1024;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    float* d_in;
    float* d_out;

    hipMalloc((void**)&d_in, ARRAY_BYTES);
    hipMalloc((void**)&d_out, ARRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    square << <1, ARRAY_SIZE >> > (d_out, d_in);

    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);


    for (int i = 0; i < ARRAY_SIZE; i++) {
        cout << setw(20) << h_out[i];
        if (i % 5 == 0) {
            cout << endl;
        }
    }

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
